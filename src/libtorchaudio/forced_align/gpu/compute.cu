#include "hip/hip_runtime.h"
#include <libtorchaudio/utils.h>
#include <torch/csrc/stable/library.h>

#include <hipcub/hipcub.hpp>
#include <limits.h>

namespace {
constexpr int kNumThreads =
    1024; // Number of threads to run CUDA kernel in parallel.
constexpr int kBackPtrBufferSize =
    100; // Buffer size of backPtr on GPU. The data is transferred to CPU once
         // the buffer reaches this max size.
} // anonymous namespace
namespace torchaudio {
namespace alignment {
namespace gpu {

using torch::stable::Tensor;
using torch::headeronly::ScalarType;

template <typename scalar_t, typename target_t>
__global__ void falign_cuda_step_kernel(
    const at::PackedTensorAccessor32<scalar_t, 3, at::RestrictPtrTraits>
        logProbs_a,
    const at::PackedTensorAccessor32<target_t, 2, at::RestrictPtrTraits>
        targets_a,
    const int T,
    const int L,
    const int N,
    const int R,
    const int t,
    const int64_t blank,
    int start,
    int end,
    int backPtrBufferLen,
    at::PackedTensorAccessor32<scalar_t, 2, at::RestrictPtrTraits>
        alphas_a,
    at::PackedTensorAccessor32<int8_t, 2, at::RestrictPtrTraits>
        backPtrBuffer_a) {
  scalar_t kNegInfinity = -std::numeric_limits<scalar_t>::infinity();
  const int batchIndex =
      0; // TODO: support batch version and use the real batch index
  int S = 2 * L + 1;
  int curIdxOffset = (t % 2); // current time step frame for alpha
  int prevIdxOffset = ((t - 1) % 2); // previous time step frame for alpha
  // reset alpha and backPtrBuffer values
  for (unsigned int i = threadIdx.x; i < S; i += blockDim.x) {
    alphas_a[curIdxOffset][i] = kNegInfinity;
    backPtrBuffer_a[backPtrBufferLen][i] = -1;
  }
  // This sync could potentially be removed through careful indexing inside each
  // thread for the above for loop. But this is okay for now.
  __syncthreads();
  if (t == 0) {
    for (unsigned int i = start + threadIdx.x; i < end; i += blockDim.x) {
      int labelIdx = (i % 2 == 0) ? blank : targets_a[batchIndex][i / 2];
      alphas_a[curIdxOffset][i] = logProbs_a[batchIndex][0][labelIdx];
    }
    return;
  }
  using BlockReduce = hipcub::BlockReduce<scalar_t, kNumThreads>;
  __shared__ typename BlockReduce::TempStorage tempStorage;
  __shared__ scalar_t maxValue;
  scalar_t threadMax;
  int startloop = start;
  threadMax = kNegInfinity;
  if (start == 0 && threadIdx.x == 0) {
    alphas_a[curIdxOffset][0] =
        alphas_a[prevIdxOffset][0] + logProbs_a[batchIndex][t][blank];
    threadMax = max(threadMax, alphas_a[curIdxOffset][0]);
    backPtrBuffer_a[backPtrBufferLen][0] = 0;
  }
  if (start == 0) {
    startloop += 1;
  }
  for (unsigned int i = startloop + threadIdx.x; i < end; i += blockDim.x) {
    scalar_t x0 = alphas_a[prevIdxOffset][i];
    scalar_t x1 = alphas_a[prevIdxOffset][i - 1];
    scalar_t x2 = kNegInfinity;
    int labelIdx = (i % 2 == 0) ? blank : targets_a[batchIndex][i / 2];
    if (i % 2 != 0 && i != 1 &&
        targets_a[batchIndex][i / 2] != targets_a[batchIndex][i / 2 - 1]) {
      x2 = alphas_a[prevIdxOffset][i - 2];
    }
    scalar_t result = 0.0;
    if (x2 > x1 && x2 > x0) {
      result = x2;
      backPtrBuffer_a[backPtrBufferLen][i] = 2;
    } else if (x1 > x0 && x1 > x2) {
      result = x1;
      backPtrBuffer_a[backPtrBufferLen][i] = 1;
    } else {
      result = x0;
      backPtrBuffer_a[backPtrBufferLen][i] = 0;
    }
    alphas_a[curIdxOffset][i] = result + logProbs_a[batchIndex][t][labelIdx];
    threadMax = max(threadMax, alphas_a[curIdxOffset][i]);
  }
#if CUDART_VERSION >= 12090  // CUDA 12.9 and later
  scalar_t maxResult = BlockReduce(tempStorage).Reduce(threadMax, thrust::maximum<scalar_t>());
#else
  scalar_t maxResult = BlockReduce(tempStorage).Reduce(threadMax, hipcub::Max());
#endif
  if (threadIdx.x == 0) {
    maxValue = maxResult;
  }
  __syncthreads();
  // normalize alpha values so that they don't overflow for large T
  for (unsigned int i = threadIdx.x; i < S; i += blockDim.x) {
    alphas_a[curIdxOffset][i] -= maxValue;
  }
}

template <typename scalar_t, ScalarType target_scalar_type>
void forced_align_impl(
    const Tensor& logProbs,
    const Tensor& targets,
    const int64_t blank,
    Tensor& paths) {
  auto defaultStream = at::cuda::getCurrentCUDAStream();
  auto cpuDataTranferStream = at::cuda::getStreamFromPool();
  const scalar_t kNegInfinity = -std::numeric_limits<scalar_t>::infinity();
  using target_t = typename std::
      conditional<target_scalar_type == ScalarType::Int, int, int64_t>::type;
  auto paths_a = paths.accessor<target_t, 2>();
  const int batchIndex =
      0; // TODO: support batch version and use the real batch index
  const int T = logProbs.size(1); // num frames
  const int N = logProbs.size(2); // alphabet size
  const int L = targets.size(1); // label length
  const int S = 2 * L + 1;

  auto targetsCpu = torch::stable::cpu(targets);
  // backPtrBuffer stores the index offset fthe best path at current position
  // We copy the values to CPU after running every kBackPtrBufferSize of
  // frames.
  Tensor backPtrBuffer = torch::stable::new_empty(logProbs, {min(kBackPtrBufferSize, T), S}, ScalarType::Char);
  torch::stable::fill_(backPtrBuffer, -1);

  Tensor backPtrCpu = torch::stable::new_empty(targetsCpu, {T, S}, ScalarType::Char);
  torch::stable::fill_(backPtrCpu, -1);

  // we store only two time frames for alphas
  // alphas for compute current timeframe can be computed only from previous
  // time frame.
  Tensor alphas = torch::stable::new_empty(logProbs, {2, S});
  torch::stable::fill_(alphas, kNegInfinity);

  // CPU accessors
  auto targetsCpu_a = targetsCpu.accessor<target_t, 2>();
  auto backPtrCpu_a = backPtrCpu.accessor<int8_t, 2>();
  // count the number of repeats in label
  int R = 0;
  for (int i = 1; i < L; ++i) {
    if (targetsCpu_a[batchIndex][i] == targetsCpu_a[batchIndex][i - 1]) {
      ++R;
    }
  }
  STD_TORCH_CHECK(
      T >= L + R,
      "targets length is too long for CTC. Found log_probs length: ",
      T,
      ", targets length: ",
      L,
      ", and number of repeats: ",
      R);
  int start = (T - (L + R)) > 0 ? 0 : 1;
  int end = (S == 1) ? 1 : 2;
  int backPtrBufferLen = 0;
  Tensor bufferCopy;
  for (int t = 0; t < T; ++t) {
    if (t > 0) {
      if (T - t <= L + R) {
        if ((start % 2 == 1) &&
            (targetsCpu_a[batchIndex][start / 2] !=
             targetsCpu_a[batchIndex][start / 2 + 1])) {
          start = start + 1;
        }
        start = start + 1;
      }
      if (t <= L + R) {
        if ((end % 2 == 0) && (end < 2 * L) &&
            (targetsCpu_a[batchIndex][end / 2 - 1] !=
             targetsCpu_a[batchIndex][end / 2])) {
          end = end + 1;
        }
        end = end + 1;
      }
    }
    falign_cuda_step_kernel<scalar_t, target_t>
        <<<1, kNumThreads, 0, defaultStream>>>(
            logProbs.packed_accessor32<scalar_t, 3, at::RestrictPtrTraits>(),
            targets.packed_accessor32<target_t, 2, at::RestrictPtrTraits>(),
            T,
            L,
            N,
            R,
            t,
            blank,
            start,
            end,
            backPtrBufferLen,
            alphas.packed_accessor32<scalar_t, 2, at::RestrictPtrTraits>(),
            backPtrBuffer
                .packed_accessor32<int8_t, 2, at::RestrictPtrTraits>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
    ++backPtrBufferLen;
    if (backPtrBufferLen == kBackPtrBufferSize || t == T - 1) {
      cpuDataTranferStream.synchronize();
      // GPU -> GPU copy
      bufferCopy = backPtrBuffer.clone();
      STD_TORCH_CHECK(bufferCopy.is_contiguous(), "unexpected fail, need to implement stable::Tensor::contiguous()")
      defaultStream.synchronize();
      at::cuda::setCurrentCUDAStream(cpuDataTranferStream);
      // Copy ASYNC from GPU to CPU
      int64_t offset =
          static_cast<int64_t>(t + 1 - backPtrBufferLen) * S * sizeof(int8_t);
      C10_CUDA_CHECK(hipMemcpyAsync(
          static_cast<int8_t*>(backPtrCpu.data_ptr()) + offset,
          bufferCopy.data_ptr(),
          backPtrBufferLen * S * sizeof(int8_t),
          hipMemcpyDeviceToHost,
          cpuDataTranferStream));
      at::cuda::setCurrentCUDAStream(defaultStream);
      backPtrBufferLen = 0;
    }
  }
  cpuDataTranferStream.synchronize();

  auto alphasCpu = torch::stable::cpu(alphas);
  auto alphasCpu_a = alphasCpu.accessor<scalar_t, 2>();
  int curIdxOffset = ((T - 1) % 2);
  int ltrIdx =
      alphasCpu_a[curIdxOffset][S - 1] > alphasCpu_a[curIdxOffset][S - 2]
      ? S - 1
      : S - 2;
  for (int t = T - 1; t >= 0; --t) {
    auto lbl_idx =
        ltrIdx % 2 == 0 ? blank : targetsCpu_a[batchIndex][ltrIdx / 2];
    paths_a[batchIndex][t] = lbl_idx;
    ltrIdx -= backPtrCpu_a[t][ltrIdx];
  }
}

std::tuple<Tensor, Tensor> compute(
    const Tensor& logProbs,
    const Tensor& targets,
    const Tensor& inputLengths,
    const Tensor& targetLengths,
    const int64_t blank) {

  STD_TORCH_CHECK(logProbs.is_cuda(), "log_probs must be a CUDA tensor");
  STD_TORCH_CHECK(targets.is_cuda(), "targets must be a CUDA tensor");
  STD_TORCH_CHECK(
      logProbs.get_device_index() == targets.get_device_index(),
      "log_probs and targets need to be on the same device");
  STD_TORCH_CHECK(inputLengths.is_cuda(), "input_lengths must be a CUDA tensor");
  STD_TORCH_CHECK(targetLengths.is_cuda(), "target_lengths must be a CUDA tensor");
  STD_TORCH_CHECK(
      logProbs.scalar_type() == ScalarType::Double ||
          logProbs.scalar_type() == ScalarType::Float ||
          logProbs.scalar_type() == ScalarType::Half,
      "log_probs must be float64, float32 or float16 (half) type");
  STD_TORCH_CHECK(
      targets.scalar_type() == ScalarType::Int || targets.scalar_type() == ScalarType::Long,
      "targets must be int32 or int64 type");
  STD_TORCH_CHECK(logProbs.is_contiguous(), "log_probs must be contiguous");
  STD_TORCH_CHECK(targets.is_contiguous(), "targets must be contiguous");
  STD_TORCH_CHECK(
      logProbs.dim() == 3,
      "log_probs must be 3-D (batch_size, input length, num classes)");
  STD_TORCH_CHECK(
      targets.dim() == 2, "targets must be 2-D (batch_size, target length,)");
  STD_TORCH_CHECK(
      inputLengths.dim() == 1, "input_lengths must be 1-D (batch_size,)");
  STD_TORCH_CHECK(
      targetLengths.dim() == 1, "target_lengths must be 1-D (batch_size,)");
  STD_TORCH_CHECK(
      logProbs.size(0) == 1,
      "The batch dimension for log_probs must be 1 at the current version.")
  STD_TORCH_CHECK(
      targets.size(0) == 1,
      "The batch dimension for targets must be 1 at the current version.")
  STD_TORCH_CHECK(
      blank >= 0 && blank < logProbs.size(-1),
      "blank must be within [0, num classes)");

  STD_TORCH_CHECK(logProbs.size(1) == torchaudio::util::max<int>(inputLengths),
      "input length mismatch");
  STD_TORCH_CHECK(
      targets.size(1) == torchaudio::util::max<int>(targetLengths),
      "target length mismatch");

  auto B = logProbs.size(0);
  auto T = logProbs.size(1); // num frames

  Tensor paths = torch::stable::new_empty(targets, {B, T}, std::nullopt, aoti_torch_device_type_cpu());
  torch::stable::zero_(paths);

  switch (logProbs.scalar_type()) {
  case ScalarType::Double: {
    if (targets.scalar_type() == ScalarType::Long) {
      forced_align_impl<double, ScalarType::Long>(logProbs, targets, blank, paths);
    } else if (targets.scalar_type() == ScalarType::Int) {
      forced_align_impl<double, ScalarType::Int>(logProbs, targets, blank, paths);
    } else {
      STD_TORCH_CHECK(false, "unreachable");
    }
    break;
  }
  case ScalarType::Float: {
    if (targets.scalar_type() == ScalarType::Long) {
      forced_align_impl<float, ScalarType::Long>(logProbs, targets, blank, paths);
    } else if (targets.scalar_type() == ScalarType::Int) {
      forced_align_impl<float, ScalarType::Int>(logProbs, targets, blank, paths);
    } else {
      STD_TORCH_CHECK(false, "unreachable");
    }
    break;
  }
  case ScalarType::Half: {
    if (targets.scalar_type() == ScalarType::Long) {
      forced_align_impl<c10::Half, ScalarType::Long>(logProbs, targets, blank, paths);
    } else if (targets.scalar_type() == ScalarType::Int) {
      forced_align_impl<c10::Half, ScalarType::Int>(logProbs, targets, blank, paths);
    } else {
      STD_TORCH_CHECK(false, "unreachable");
    }
    break;
  }
  default: {
    STD_TORCH_CHECK(false, "unreachable");
  }
  };
  Tensor pathsCuda = torch::stable::new_empty(paths,
                                              torchaudio::util::sizes(paths),
                                              std::nullopt,
                                              aoti_torch_device_type_cuda(),
                                              logProbs.get_device_index());
  torch::stable::copy_(pathsCuda, paths);
  return std::make_tuple(pathsCuda, logProbs);
}

void boxed_forced_align_gpu(StableIValue* stack, uint64_t num_args, uint64_t num_outputs) {
  STD_TORCH_CHECK(num_args == 5, "num_args must be 5");
  STD_TORCH_CHECK(num_outputs == 2, "num_outputs must be 2");
  std::tuple<Tensor, Tensor> res = compute(
      /*logProbs*/to<Tensor>(stack[0]),
      /*targets*/to<Tensor>(stack[1]),
      /*logit_lengths*/to<Tensor>(stack[2]),
      /*target_lengths*/to<Tensor>(stack[3]),
      /*blank*/float(to<int64_t>(stack[4])));
  stack[0] = from(std::get<0>(res));
  stack[1] = from(std::get<1>(res));
}

STABLE_TORCH_LIBRARY_IMPL(torchaudio, CUDA, m) {
  m.impl("forced_align", &boxed_forced_align_gpu);
}

} // namespace gpu
} // namespace alignment
} // namespace torchaudio
