#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAStream.h>
#include <torch/types.h>
#include <torchaudio/csrc/rnnt/gpu/gpu_transducer.h>

namespace torchaudio {
namespace rnnt {
namespace gpu {

// Entry point into RNNT Loss
std::tuple<torch::Tensor, c10::optional<torch::Tensor>> compute(
    torch::Tensor& logits,
    const torch::Tensor& targets,
    const torch::Tensor& logit_lengths,
    const torch::Tensor& target_lengths,
    int64_t blank,
    double clamp,
    bool fused_log_softmax = true) {
  TORCH_CHECK(
      logits.device().type() == targets.device().type(),
      "logits and targets must be on the same device");
  TORCH_CHECK(
      logits.device().type() == logit_lengths.device().type(),
      "logits and logit_lengths must be on the same device");
  TORCH_CHECK(
      logits.device().type() == target_lengths.device().type(),
      "logits and target_lengths must be on the same device");

  TORCH_CHECK(
      logits.dtype() == torch::kFloat32 || logits.dtype() == torch::kFloat16,
      "logits must be float32 or float16 (half) type");
  TORCH_CHECK(targets.dtype() == torch::kInt32, "targets must be int32 type");
  TORCH_CHECK(
      logit_lengths.dtype() == torch::kInt32,
      "logit_lengths must be int32 type");
  TORCH_CHECK(
      target_lengths.dtype() == torch::kInt32,
      "target_lengths must be int32 type");

  TORCH_CHECK(logits.is_contiguous(), "logits must be contiguous");
  TORCH_CHECK(targets.is_contiguous(), "targets must be contiguous");
  TORCH_CHECK(
      logit_lengths.is_contiguous(), "logit_lengths must be contiguous");
  TORCH_CHECK(
      target_lengths.is_contiguous(), "target_lengths must be contiguous");

  TORCH_CHECK(
      logits.dim() == 4, "logits must be 4-D (batch, time, target, class)");
  TORCH_CHECK(
      targets.dim() == 2, "targets must be 2-D (batch, max target length)");
  TORCH_CHECK(logit_lengths.dim() == 1, "logit_lengths must be 1-D");
  TORCH_CHECK(target_lengths.dim() == 1, "target_lengths must be 1-D");

  TORCH_CHECK(
      logit_lengths.size(0) == logits.size(0),
      "batch dimension mismatch between logits and logit_lengths");
  TORCH_CHECK(
      target_lengths.size(0) == logits.size(0),
      "batch dimension mismatch between logits and target_lengths");
  TORCH_CHECK(
      targets.size(0) == logits.size(0),
      "batch dimension mismatch between logits and targets");

  TORCH_CHECK(
      blank >= 0 && blank < logits.size(-1),
      "blank must be within [0, logits.shape[-1])");

  TORCH_CHECK(
      logits.size(1) == at::max(logit_lengths).item().toInt(),
      "input length mismatch");
  TORCH_CHECK(
      logits.size(2) == at::max(target_lengths).item().toInt() + 1,
      "output length mismatch");
  TORCH_CHECK(
      targets.size(1) == at::max(target_lengths).item().toInt(),
      "target length mismatch");

  Options options;
  options.batchSize_ = logit_lengths.size(0);
  options.nHypos_ = target_lengths.size(0) / logit_lengths.size(0);
  options.maxSrcLen_ = logits.size(1);
  options.maxTgtLen_ = logits.size(2);
  options.numTargets_ = logits.size(3);
  options.blank_ = blank;
  options.clamp_ = clamp;
  options.fusedLogSmax_ = fused_log_softmax;

  TORCH_CHECK_EQ(logits.device().type(), torch::DeviceType::CUDA);
  options.stream_ = at::cuda::getCurrentCUDAStream();
  hipSetDevice(logits.get_device());
  options.device_ = GPU;

  torch::Tensor costs = torch::empty(
      options.batchSize_ * options.nHypos_,
      torch::TensorOptions().device(logits.device()).dtype(logits.dtype()));
  c10::optional<torch::Tensor> gradients = torch::zeros_like(logits);

  torch::Tensor int_workspace = torch::empty(
      IntWorkspace::ComputeSizeFromOptions(options),
      torch::TensorOptions()
          .device(logits.device())
          .dtype(torch::ScalarType::Int));

  torch::Tensor float_workspace = torch::empty(
      DtypeWorkspace<float>::ComputeSizeFromOptions(options),
      torch::TensorOptions()
          .device(logits.device())
          .dtype(torch::ScalarType::Float));

  Workspace<float> workspace(
      /*options=*/options,
      /*dtype_data=*/float_workspace.data_ptr<float>(),
      /*dtype_size=*/float_workspace.numel(),
      /*int_data=*/int_workspace.data_ptr<int>(),
      /*int_size=*/int_workspace.numel());

  switch (logits.scalar_type()) {
    case torch::ScalarType::Float: {
      Compute</*DTYPE=*/float, /*CAST_DTYPE=*/float>(
          /*workspace=*/workspace,
          /*logits=*/logits.data_ptr<float>(),
          /*targets=*/targets.data_ptr<int>(),
          /*logit_lengths=*/logit_lengths.data_ptr<int>(),
          /*target_lengths=*/target_lengths.data_ptr<int>(),
          /*costs=*/costs.data_ptr<float>(),
          /*gradients=*/gradients->data_ptr<float>());
      break;
    }
    case torch::ScalarType::Half: {
      Compute</*DTYPE=*/c10::Half, /*CAST_DTYPE=*/float>(
          /*workspace=*/workspace,
          /*logits=*/logits.data_ptr<c10::Half>(),
          /*targets=*/targets.data_ptr<int>(),
          /*logit_lengths=*/logit_lengths.data_ptr<int>(),
          /*target_lengths=*/target_lengths.data_ptr<int>(),
          /*costs=*/costs.data_ptr<c10::Half>(),
          /*gradients=*/gradients->data_ptr<c10::Half>());
      break;
    }
    default: {
      break;
    }
  };

  return std::make_tuple(costs, gradients);
}

TORCH_LIBRARY_IMPL(torchaudio, CUDA, m) {
  m.impl("rnnt_loss", &compute);
}

} // namespace gpu
} // namespace rnnt
} // namespace torchaudio
