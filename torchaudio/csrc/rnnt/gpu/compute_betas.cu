#include <c10/cuda/CUDAStream.h>
#include <torch/script.h>
#include <torchaudio/csrc/rnnt/gpu/gpu_transducer.h>

namespace torchaudio {
namespace rnnt {
namespace gpu {

torch::Tensor compute_betas(
    const torch::Tensor& logits,
    const torch::Tensor& targets,
    const torch::Tensor& src_lengths,
    const torch::Tensor& tgt_lengths,
    int64_t blank,
    double clamp) {
  Options options;
  options.batchSize_ = src_lengths.size(0);
  options.nHypos_ = tgt_lengths.size(0) / src_lengths.size(0);
  options.maxSrcLen_ = logits.size(1);
  options.maxTgtLen_ = logits.size(2);
  options.numTargets_ = logits.size(3);
  options.blank_ = blank;
  options.clamp_ = clamp;

  CHECK_EQ(logits.device().type(), torch::DeviceType::CUDA);
  options.stream_ = at::cuda::getCurrentCUDAStream();
  hipSetDevice(logits.get_device());
  options.device_ = GPU;

  torch::Tensor costs = torch::empty(
      tgt_lengths.size(0),
      torch::TensorOptions().device(logits.device()).dtype(logits.dtype()));

  torch::Tensor betas = torch::zeros(
      {options.batchSize_ * options.nHypos_,
       options.maxSrcLen_,
       options.maxTgtLen_},
      torch::TensorOptions().device(logits.device()).dtype(logits.dtype()));

  torch::Tensor int_workspace = torch::empty(
      IntWorkspace::ComputeSizeFromOptions(options),
      torch::TensorOptions()
          .device(logits.device())
          .dtype(torch::ScalarType::Int));

  torch::Tensor float_workspace = torch::empty(
      DtypeWorkspace<float>::ComputeSizeFromOptions(options),
      torch::TensorOptions()
          .device(logits.device())
          .dtype(torch::ScalarType::Float));

  Workspace<float> workspace(
      /*options=*/options,
      /*dtype_data=*/float_workspace.data_ptr<float>(),
      /*dtype_size=*/float_workspace.numel(),
      /*int_data=*/int_workspace.data_ptr<int>(),
      /*int_size=*/int_workspace.numel());

  // Only support float, this is mainly to enable easy
  // unit-testing
  ComputeBetas</*DTYPE=*/float, /*CAST_DTYPE=*/float>(
      /*workspace=*/workspace,
      /*logits=*/logits.data_ptr<float>(),
      /*targets=*/targets.data_ptr<int>(),
      /*src_lengths=*/src_lengths.data_ptr<int>(),
      /*tgt_lengths=*/tgt_lengths.data_ptr<int>(),
      /*costs=*/costs.data_ptr<float>(),
      /*betas=*/betas.data_ptr<float>());
  return betas;
}

TORCH_LIBRARY_IMPL(torchaudio, CUDA, m) {
  m.impl("rnnt_loss_betas", &compute_betas);
}

} // namespace gpu
} // namespace rnnt
} // namespace torchaudio
